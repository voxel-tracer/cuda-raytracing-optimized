#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "rnd.h"
#include "vec3.h"
#include "camera.h"
#include "triangle.h"
#include "material.h"

#define STATS

const int kMaxTris = 600;
__device__ __constant__ vec3 d_triangles[kMaxTris * 3];

#ifdef STATS
#define NUM_RAYS_PRIMARY            0
#define NUM_RAYS_SECONDARY          1
#define NUM_RAYS_SHADOWS            2
#define NUM_RAYS_LOW_POWER          3
#define NUM_RAYS_PRIMARY_NO_HITS    4
#define NUM_RAYS_SIZE               5
#endif

struct RenderContext {
    vec3* fb;
    uint16_t numTris;
    int nx;
    int ny;
    int ns;
    camera cam;
    material* materials;
    uint16_t numMats;
    float* hdri = NULL;
#ifdef STATS
    uint32_t* numRays;
#endif
};

RenderContext renderContext;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool hit(const ray& r, uint16_t numTris, float t_min, float t_max, hit_record& rec, bool isShadow) {
    hit_record temp_rec;
    bool hit_anything = false;
    float closest_so_far = t_max;
    for (int i = 0; i < numTris; i++) {
        if (triangleHit(d_triangles + i * 3, r, t_min, closest_so_far, temp_rec)) {
            if (isShadow) return true;

            hit_anything = true;
            closest_so_far = temp_rec.t;
            rec = temp_rec;
            rec.hitIdx = i;
        }
    }
    rec.hitIdx = rec.hitIdx < (numTris - 2) ? 0 : 1; // last 2 triangles is for the floor
    return hit_anything;
}

__device__ bool generateShadowRay(const hit_record& hit, ray& shadow, vec3& emitted, rand_state& state) {
    const vec3 lightCenter(-2000, 0, 5000);
    const float lightRadius = 500;
    const vec3 lightColor = vec3(1, 1, 1) * 100;

    // create a random direction towards the light
    // coord system for sampling
    const vec3 sw = unit_vector(lightCenter - hit.p);
    const vec3 su = unit_vector(cross(fabs(sw.x()) > 0.01f ? vec3(0, 1, 0) : vec3(1, 0, 0), sw));
    const vec3 sv = cross(sw, su);

    // sample sphere by solid angle
    const float cosAMax = sqrt(1.0f - lightRadius * lightRadius / (hit.p - lightCenter).squared_length());
    const float eps1 = rnd(state);
    const float eps2 = rnd(state);
    const float cosA = 1.0f - eps1 + eps1 * cosAMax;
    const float sinA = sqrt(1.0f - cosA * cosA);
    const float phi = 2 * M_PI * eps2;
    const vec3 l = unit_vector(su * cosf(phi) * sinA + sv * sinf(phi) * sinA + sw * cosA);

    const float dotl = dot(l, hit.normal);
    if (dotl <= 0)
        return false;

    const float omega = 2 * M_PI * (1.0f - cosAMax);
    shadow = ray(hit.p, l);
    emitted = lightColor * dotl * omega / M_PI;

    return true;
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, const RenderContext& context, rand_state& state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    vec3 curColor = vec3(0, 0, 0);
    for (int bounce = 0; bounce < 50; bounce++) {
#ifdef STATS
        if (bounce == 0)
            atomicAdd(context.numRays + NUM_RAYS_PRIMARY, 1);
        else
            atomicAdd(context.numRays + NUM_RAYS_SECONDARY, 1);
        if (cur_attenuation.length() < 0.01f)
            atomicAdd(context.numRays + NUM_RAYS_LOW_POWER, 1);
#endif
        hit_record rec;
        if (hit(cur_ray, context.numTris, 0.001f, FLT_MAX, rec, false)) {
#ifdef STATS
            if (bounce == 0 && rec.hitIdx == 1) // primary ray hit didn't hit the main model
                atomicAdd(context.numRays + NUM_RAYS_PRIMARY_NO_HITS, 1);
#endif
            ray scattered;
            vec3 attenuation;
            bool hasShadow;
            if (scatter(context.materials[rec.hitIdx], cur_ray, rec, attenuation, scattered, state, hasShadow)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;

                // trace shadow ray if needed
                ray shadow;
                vec3 emitted;
                if (hasShadow && generateShadowRay(rec, shadow, emitted, state)) {
#ifdef STATS
                    atomicAdd(context.numRays + NUM_RAYS_SHADOWS, 1);
#endif
                    if (!hit(shadow, context.numTris, 0.001f, FLT_MAX, rec, true)) {
                        // intersection point is illuminated by the light
                        curColor += emitted * cur_attenuation;
                    }
                }
            }
            else {
                return curColor;
            }
        }
        else {
#ifdef STATS
            if (bounce == 0) // primary ray hit didn't hit anything
                atomicAdd(context.numRays + NUM_RAYS_PRIMARY_NO_HITS, 1);
#endif
            if (context.hdri != NULL) {
                // environment map
                vec3 dir = unit_vector(cur_ray.direction());
                uint2 coords = make_uint2(-atan2(dir.x(), dir.y()) * 1024 / (2 * M_PI), acos(dir.z()) * 512 / M_PI);
                vec3 c(
                    context.hdri[(coords.y * 1024 + coords.x)*3],
                    context.hdri[(coords.y * 1024 + coords.x)*3 + 1],
                    context.hdri[(coords.y * 1024 + coords.x)*3 + 2]
                );
                return cur_attenuation * c;
            }
            else {
                // sky color
                vec3 unit_direction = cur_ray.direction();
                float t = 0.5f * (unit_direction.z() + 1.0f);
                vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
                curColor += c * cur_attenuation;
                return curColor;
                // uniform sky color
                //curColor += cur_attenuation * .5f;
                //return curColor;
            }
        }
    }
    return curColor; // exceeded recursion
}

__global__ void render(const RenderContext context) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= context.nx) || (j >= context.ny)) return;
    int pixel_index = j * context.nx + i;
    rand_state state = (wang_hash(pixel_index) * 336343633) | 1;

    vec3 col(0, 0, 0);
    for (int s = 0; s < context.ns; s++) {
        float u = float(i + rnd(state)) / float(context.nx);
        float v = float(j + rnd(state)) / float(context.ny);
        ray r = get_ray(context.cam, u, v, state);
        col += color(r, context, state);
    }
    col /= float(context.ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    context.fb[pixel_index] = col;
}

extern "C" void
initRenderer(const vec3 *h_triangles, uint16_t numTris, material* h_materials, uint16_t numMats, const camera cam, vec3 **fb, int nx, int ny) {
    renderContext.nx = nx;
    renderContext.ny = ny;

    size_t fb_size = nx * ny * sizeof(vec3);
    checkCudaErrors(hipMallocManaged((void**)&(renderContext.fb), fb_size));
    *fb = renderContext.fb;

    // all triangles share the same material
    checkCudaErrors(hipMalloc((void**)&renderContext.materials, numMats * sizeof(material)));
    checkCudaErrors(hipMemcpy(renderContext.materials, h_materials, numMats * sizeof(material), hipMemcpyHostToDevice));
    renderContext.numMats = numMats;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_triangles), h_triangles, numTris * 3 * sizeof(vec3)));
    renderContext.numTris = numTris;

    renderContext.cam = cam;
#ifdef STATS
    checkCudaErrors(hipMallocManaged((void**)&(renderContext.numRays), NUM_RAYS_SIZE * sizeof(uint32_t)));
    memset(renderContext.numRays, 0, NUM_RAYS_SIZE * sizeof(uint32_t));
#endif
}

extern "C" 
void initHDRi(float* data, int x, int y, int n) {
    checkCudaErrors(hipMalloc((void**)&renderContext.hdri, x * y * n * sizeof(float)));
    checkCudaErrors(hipMemcpy(renderContext.hdri, data, x * y * n * sizeof(float), hipMemcpyHostToDevice));
}

extern "C" void
runRenderer(int ns, int tx, int ty) {
    renderContext.ns = ns;

    // Render our buffer
    dim3 blocks(renderContext.nx / tx + 1, renderContext.ny / ty + 1);
    dim3 threads(tx, ty);
    render <<<blocks, threads >>> (renderContext);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
#ifdef STATS
    std::cerr << "num rays:\n";
    std::cerr << "\tprimary:\t" << renderContext.numRays[NUM_RAYS_PRIMARY] << std::endl;
    std::cerr << "\tprimary (no hit):\t" << renderContext.numRays[NUM_RAYS_PRIMARY_NO_HITS] << std::endl;
    std::cerr << "\tsecondary:\t" << renderContext.numRays[NUM_RAYS_SECONDARY] << std::endl;
    std::cerr << "\tshadows:\t" << renderContext.numRays[NUM_RAYS_SHADOWS] << std::endl;
    std::cerr << "\tpower < 0.1:\t" << renderContext.numRays[NUM_RAYS_LOW_POWER] << std::endl;
#endif
}

extern "C" void
cleanupRenderer() {
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(renderContext.materials));
    checkCudaErrors(hipFree(renderContext.fb));
    if (renderContext.hdri != NULL) checkCudaErrors(hipFree(renderContext.hdri));

    hipDeviceReset();
}